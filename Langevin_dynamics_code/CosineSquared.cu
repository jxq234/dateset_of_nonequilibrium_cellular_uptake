#include "hip/hip_runtime.h"
/*
GALAMOST - GPU-Accelerated Large-Scale Molecular Simulation Toolkit
COPYRIGHT
	GALAMOST Copyright (c) (2013) The group of Prof. Zhong-Yuan Lu
LICENSE
	This program is a free software: you can redistribute it and/or
	modify it under the terms of the GNU General Public License.
	This program is distributed in the hope that it will be useful,
	but WITHOUT ANY WARRANTY; without even the implied warranty of
	MERCHANT ABILITY or FITNESS FOR A PARTICULAR PURPOSE.
	See the General Public License v3 for more details.
	You should have received a copy of the GNU General Public License
	along with this program. If not, see <http://www.gnu.org/licenses/>.
DISCLAIMER
	The authors of GALAMOST do not guarantee that this program and its
	derivatives are free from error. In no event shall the copyright
	holder or contributors be liable for any indirect, incidental,
	special, exemplary, or consequential loss or damage that results
	from its use. We also have no responsibility for providing the
	service of functional extension of this program to general users.
USER OBLIGATION
	If any results obtained with GALAMOST are published in the scientific
	literature, the users have an obligation to distribute this program
	and acknowledge our efforts by citing the paper "Y.-L. Zhu, H. Liu,
	Z.-W. Li, H.-J. Qian, G. Milano, and Z.-Y. Lu, J. Comput. Chem. 2013,
	34, 2197-2211" in their article.
CORRESPONDENCE
	State Key Laboratory of Polymer Physics and Chemistry,
	Changchun Institute of Applied Chemistry, Chinese Academy of Sciences, China,
	Dr. You-Liang Zhu,
	Email: youliangzhu@ciac.ac.cn
*/
//	Maintainer: You-Liang Zhu

#include "CosineSquared.cuh"


Real4_tex_t pos_tex;
__global__ void gpu_compute_cosine_squared_kernel(Real4* d_force,
	ForceLog force_log,
	Real4* d_pos,
	BoxSize box,
	const unsigned int* d_n_neigh,
	const unsigned int* d_nlist,
	Index2D nli,
	Real4* d_params,
	int coeff_width,
	unsigned int Np,
	bool energy_shift)
{
	extern __shared__ Real4 s_params[];
	for (unsigned int cur_offset = 0; cur_offset < coeff_width * coeff_width; cur_offset += blockDim.x)
	{
		if (cur_offset + threadIdx.x < coeff_width * coeff_width)
			s_params[cur_offset + threadIdx.x] = d_params[cur_offset + threadIdx.x];
	}
	__syncthreads();

	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= Np)
		return;

	unsigned int n_neigh = d_n_neigh[idx];
	Real4 pos = texFetchReal4(d_pos, pos_tex, idx);

	Real4 force = d_force[idx];
	Real virial = Real(0.0);
	Real6 virial_matrix = ToReal6(0.0, 0.0, 0.0, 0.0, 0.0, 0.0);
	if (force_log.virial)
		virial = force_log.d_virial[idx];
	if (force_log.virial_matrix)
		virial_matrix = force_log.d_virial_matrix[idx];

	unsigned int cur_neigh = 0;
	unsigned int next_neigh = d_nlist[nli(idx, 0)];

	for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
	{
		cur_neigh = next_neigh;
		next_neigh = d_nlist[nli(idx, neigh_idx + 1)];
		Real4 neigh_pos = texFetchReal4(d_pos, pos_tex, cur_neigh);

		Real dx = pos.x - neigh_pos.x;
		Real dy = pos.y - neigh_pos.y;
		Real dz = pos.z - neigh_pos.z;

		box.minDisImage(dx, dy, dz);
		Real rsq = dx * dx + dy * dy + dz * dz;
        Real r = sqrt_gala(rsq);
		int typ_pair = __real_as_int(neigh_pos.w) * coeff_width + __real_as_int(pos.w);

		Real epsilon = s_params[typ_pair].x;
		Real rcut = s_params[typ_pair].y;
		Real rcutsq = s_params[typ_pair].z;
		Real wc = s_params[typ_pair].w;
		Real wcsq = wc * wc;

		if (r < rcut)
		{

			Real pair_eng = -epsilon;
			force.w += pair_eng * Real(0.5);
		}

		if (r >= rcut && r <= (rcut+wc)  )
		{
			Real rinv = Real(1.0) / r;
			// Real r6inv = r2inv * r2inv * r2inv;
			// Real force_divr = r2inv * r6inv * (Real(12.0) * lj1 * r6inv - Real(6.0) * lj2);

			if (energy_shift)
            {
                
            }
			Real piw = Real(M_PI) / Real(2.0) / wc;
			Real deltarpiw = piw * (r - rcut);
            Real cosdeltarpiw = cos_gala(deltarpiw);
			Real force_divr = - 2 * epsilon * cosdeltarpiw * sin_gala(deltarpiw) * piw * rinv;

			//Real pair_eng = r6inv * (lj1 * r6inv - lj2);
			Real pair_eng = - epsilon * cosdeltarpiw * cosdeltarpiw;


			if (force_log.virial)
				virial += Real(1.0) / Real(6.0) * rsq * force_divr;
			if (force_log.virial_matrix)
			{
                Real force_div2r = Real(0.5) * force_divr;
				virial_matrix.x += dx * dx * force_div2r;   // press_tensor_xx
				virial_matrix.y += dx * dy * force_div2r;   // press_tensor_xy
				virial_matrix.z += dx * dz * force_div2r;   // press_tensor_xz
				virial_matrix.w += dy * dy * force_div2r;   // press_tensor_yy
				virial_matrix.m += dy * dz * force_div2r;   // press_tensor_yz
				virial_matrix.n += dz * dz * force_div2r;   // press_tensor_zz			
			}

			force.x += dx * force_divr;
			force.y += dy * force_divr;
			force.z += dz * force_divr;
			// energy is double counted: multiply by 0.5		
			force.w += pair_eng * Real(0.5);
		}
	}

	d_force[idx] = force;
	if (force_log.virial)
		force_log.d_virial[idx] = virial;
	if (force_log.virial_matrix)
		force_log.d_virial_matrix[idx] = virial_matrix;
}


hipError_t gpu_compute_cosine_squared(Real4* d_force,
	ForceLog& force_log,
	Real4* d_pos,
	const BoxSize& box,
	const unsigned int* d_n_neigh,
	const unsigned int* d_nlist,
	const Index2D& nli,
	Real4* d_params,
	int coeff_width,
	int blocksize,
	unsigned int Np,
	unsigned int Ntot,
	bool energy_shift,
	unsigned int compute_capability)
{
	dim3 grid((int)ceil((Real)Np / (Real)blocksize), 1, 1);
	dim3 threads(blocksize, 1, 1);

	if (compute_capability < 350)
	{
		pos_tex.normalized = false;
		pos_tex.filterMode = hipFilterModePoint;
		hipError_t error = hipBindTexture(0, pos_tex, d_pos, sizeof(Real4) * Ntot);
		if (error != hipSuccess)
			return error;
	}

	gpu_compute_cosine_squared_kernel << < grid, threads, sizeof(Real4)* coeff_width* coeff_width >> > (d_force,
		force_log,
		d_pos,
		box,
		d_n_neigh,
		d_nlist,
		nli,
		d_params,
		coeff_width,
		Np,
		energy_shift);


	return hipSuccess;
}


